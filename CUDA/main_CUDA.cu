#include "hip/hip_runtime.h"
#include<iostream>
#include<fstream>
#include<string>
#include<algorithm>
#include<vector>
#include<stdio.h>
#include<stdlib.h>
#include<math.h>
#include<limits>
#include<iomanip>
#include <chrono>
#include "input_image.h"
#include "complex.h"

#define BLOCK_SIZE 16
using namespace std;
const float PI = 3.14159265358979f;

class MyComplex{
	public:
		__device__ __host__ MyComplex() : real(0.0f), imag(0.0f) {} 
		__device__ __host__ MyComplex(float r, float i) : real(r), imag(i) {}
		__device__ __host__ MyComplex operator+(const MyComplex &b) const 
		{
		MyComplex a;
		a.real = real + b.real;
		a.imag = imag + b.imag;
		return a;
		}
		__device__ __host__ MyComplex operator*(const MyComplex &b) const 
		{
		MyComplex a;
		a.real=real*b.real-imag*b.imag;
		a.imag=imag*b.real+real*b.imag;
		return a;
		}
		float real;
		float imag;
	};
std::ostream& operator<< (std::ostream& os, const MyComplex& rhs) 
{
	MyComplex c(rhs);
	if(fabsf(rhs.imag) < 1e-10) c.imag = 0.0f;
	if(fabsf(rhs.real) < 1e-10) c.real = 0.0f;

	if(c.imag == 0) os << c.real;
	else os << "(" << c.real << "," << c.imag << ")";
	return os;
}

__global__ void matrix1DFFT(MyComplex *MatOld,MyComplex *MatNew,MyComplex *Temp,int Rows,int Cols,int Operation,int direction)
{
	int j = blockIdx.x * blockDim.x + threadIdx.x;
	int i = blockIdx.y * blockDim.y + threadIdx.y;
	if (i<Cols && j <Rows)
	{
		if (direction==0)  //Forward
		{
			if (Operation==0) // operating on rows
			{
				MyComplex sum=MyComplex(0, 0);
				float theta=2*PI/Cols*j;
				MyComplex twiddle=MyComplex(0,0);			
				for (int jj = 0; jj < Cols; jj++)
				{
					twiddle=MyComplex(cos(theta*jj),sin(-theta*jj));
					sum=sum+MatOld[i*Cols+jj]*twiddle;
			 	}
			 	Temp[i*Cols+j]=sum;
			}
			else // operating on columns
			{
				MyComplex sum=MyComplex(0, 0);
				float theta=2*PI/Rows*i;
				MyComplex twiddle=MyComplex(0,0);			
				for (int ii = 0; ii < Rows; ii++)
				{
					twiddle=MyComplex(cos(theta*ii),sin(-theta*ii));
					sum=sum+Temp[ii*Cols+j]*twiddle;
			 	}
			 	MatNew[i*Cols+j]=sum;
			}
		}
		else // Reverse
		{
			if (Operation==0) // operating on rows
			{
				MyComplex sum=MyComplex(0, 0);
				float theta=2*PI/Cols*j;
				MyComplex twiddle=MyComplex(0,0);			
				for (int jj = 0; jj < Cols; jj++)
				{
					twiddle=MyComplex(cos(theta*jj),sin(theta*jj));
					sum=sum+MatOld[i*Cols+jj]*twiddle;
			 	}
			 	Temp[i*Cols+j]=sum*MyComplex(((float)1/Cols),0);
			}
			else // operating on columns
			{
				MyComplex sum=MyComplex(0, 0);
				float theta=2*PI/Rows*i;
				MyComplex twiddle=MyComplex(0,0);			
				for (int ii = 0; ii < Rows; ii++)
				{
					twiddle=MyComplex(cos(theta*ii),sin(theta*ii));
					sum=sum+Temp[ii*Cols+j]*twiddle;
			 	}
			 	MatNew[i*Cols+j]=sum*MyComplex(((float)1/Cols),0);
			}
		}	
	}
}
int main(int argc, char const *argv[])
{

	InputImage im=InputImage(argv[2]);
	int Rows=im.get_height();
	int Cols=im.get_width();

	string direction_string;
	direction_string=argv[1];
	string f ("forward");
	int direction =0;
	for (int i = 0; i < direction_string.length(); i++)
	{
		if (int(direction_string.at(i))!=int(f.at(i)))
		{
			direction=1;
		}
	}
	cout<<"Direction : "<<direction<<endl;
	
	std::chrono::time_point<std::chrono::system_clock> start, end;
	start = std::chrono::system_clock::now();

	MyComplex *d_MatOld,*d_MatNew,*d_Temp;
	int matSize= Rows*Cols*2*sizeof(float);
	hipMalloc((void **)&d_MatOld,matSize);
	hipMalloc((void **)&d_MatNew,matSize);
	hipMalloc((void **)&d_Temp,matSize);
	hipMemcpy(d_MatOld,im.get_image_data(),matSize,hipMemcpyHostToDevice);

	dim3 Block(BLOCK_SIZE,BLOCK_SIZE);
	dim3 Grid(ceil((Rows+BLOCK_SIZE-1)/BLOCK_SIZE),ceil((Cols+BLOCK_SIZE-1)/BLOCK_SIZE));
	if (direction==0)
	{
		matrix1DFFT<<<Grid,Block>>>(d_MatOld,d_MatNew,d_Temp,Rows,Cols,0,direction);
		matrix1DFFT<<<Grid,Block>>>(d_MatOld,d_MatNew,d_Temp,Rows,Cols,1,direction);
	}
	else
	{
		matrix1DFFT<<<Grid,Block>>>(d_MatOld,d_MatNew,d_Temp,Rows,Cols,0,direction);
		matrix1DFFT<<<Grid,Block>>>(d_MatOld,d_MatNew,d_Temp,Rows,Cols,1,direction);
	}
	
	hipMemcpy(im.get_image_data(),d_MatNew,matSize,hipMemcpyDeviceToHost);
	hipFree(d_Temp);hipFree(d_MatNew);hipFree(d_MatOld);
	end = std::chrono::system_clock::now();
	std::chrono::duration<double> elapsed_seconds = end - start; 
    std::time_t end_time = std::chrono::system_clock::to_time_t(end); 
    std::cout << "finished computation at " << std::ctime(&end_time) 
              << "elapsed time: " << elapsed_seconds.count() << "s\n";

    if (direction==0)
    {
    	im.save_image_data(argv[3], im.get_image_data(), Cols, Rows);
    }
    else
    {
    	im.save_image_data_real(argv[3], im.get_image_data(), Cols, Rows);
    }
	return 0;
}
